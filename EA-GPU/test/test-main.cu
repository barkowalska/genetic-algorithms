#include"CEA.cuh"
#include"Selection/TournamentSelection.cuh"

//potrzebny pomocniczy wskaznik na funkcje device bo dopiero wartosc wskaznika mozemy skopiowac do hosta
__device__ double fitness(double*){
    return 0;
}

__device__ cea::fitnessFunction_ptr myFitenss = fitness;

int main()
{
    
    double (*device_fitness_ptr)(double*);
    hipError_t err = hipMemcpyFromSymbol(&device_fitness_ptr, HIP_SYMBOL(myFitenss), sizeof(device_fitness_ptr));
    if (err != hipSuccess) {
        std::cout<<"FAILURE";
        return -1;
    }
    cea::CEA<1,10,10> ea(device_fitness_ptr);
    cea::TournamentSelection<1,20> tournamentSelection_({1,1,1}, 2);
    tournamentSelection_.printData();
}
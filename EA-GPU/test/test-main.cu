#include"CEA.cuh"

__device__ double fitness(double*){
    return 0;
}

int main()
{
    
    double (*device_fitness_ptr)(double*);
    hipError_t err = hipMemcpyFromSymbol(&device_fitness_ptr, HIP_SYMBOL(fitness), sizeof(device_fitness_ptr));
    if (err != hipSuccess) {
        // Handle error
        return -1;
    }
    cea::CEA<1,10,10> ea(device_fitness_ptr);
}
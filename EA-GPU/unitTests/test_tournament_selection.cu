// unitTests/test_tournament_selection.cu

#include <gtest/gtest.h>
#include "CEA.cuh"
#include "Selection/TournamentSelection.cuh"
#include <hip/hip_runtime.h>

using namespace cea;

// Constants for testing
const uint64_t PopSize = 8;
const uint64_t ChromosomeSize = 5;
const uint64_t TournamentSize = 3;

// Device fitness function (simple sum of chromosome elements)
__device__ double fitnessFunction(double* chromosome) {
    double fitness = 0.0;
    for (uint64_t i = 0; i < ChromosomeSize; ++i) {
        fitness += chromosome[i];
    }
    return fitness;
}

class TournamentSelectionTest : public ::testing::Test {
protected:
    PopulationType<PopSize, ChromosomeSize>* d_population = nullptr;
    uint64_t* d_selected = nullptr;

    void SetUp() override {
        // Initialize host population with known fitness values
        PopulationType<PopSize, ChromosomeSize> h_population;
        for (uint64_t i = 0; i < PopSize; ++i) {
            // Initialize chromosomes with known values
            for (uint64_t j = 0; j < ChromosomeSize; ++j) {
                h_population.chromosomes[i * ChromosomeSize + j] = static_cast<double>(i + j);
            }
            // Precompute fitness values on host
            h_population.fitnessValue[i] = 0.0;
            for (uint64_t j = 0; j < ChromosomeSize; ++j) {
                h_population.fitnessValue[i] += h_population.chromosomes[i * ChromosomeSize + j];
            }
        }

        // Allocate device memory
        hipMalloc(&d_population, sizeof(PopulationType<PopSize, ChromosomeSize>));
        hipMalloc(&d_selected, PopSize * sizeof(uint64_t));

        // Copy population to device
        hipMemcpy(d_population, &h_population, sizeof(PopulationType<PopSize, ChromosomeSize>), hipMemcpyHostToDevice);

        // Set the fitness function on the device
        fitnessFunction_ptr h_fitnessFunction;
        hipMemcpyFromSymbol(&h_fitnessFunction, HIP_SYMBOL(fitnessFunction), sizeof(fitnessFunction_ptr));
        hipMemcpyToSymbol(HIP_SYMBOL(FitnessFunction), &h_fitnessFunction, sizeof(fitnessFunction_ptr));
    }

    void TearDown() override {
        // Free device memory
        if (d_population) hipFree(d_population);
        if (d_selected) hipFree(d_selected);
    }
};

TEST_F(TournamentSelectionTest, TournamentSelectionWorksCorrectly) {
    // Set a fixed seed for reproducibility
    setGlobalSeed();

    // Create instance of TournamentSelection
    dim3 blockSize(PopSize);
    TournamentSelection<PopSize, ChromosomeSize> selection(blockSize, TournamentSize);

    // Run the selection operator
    selection(d_population, d_selected);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy selected indices back to host
    uint64_t h_selected[PopSize];
    hipMemcpy(h_selected, d_selected, PopSize * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Copy population back to host for validation
    PopulationType<PopSize, ChromosomeSize> h_population;
    hipMemcpy(&h_population, d_population, sizeof(PopulationType<PopSize, ChromosomeSize>), hipMemcpyDeviceToHost);

    // Verify the results
    for (uint64_t i = 0; i < PopSize; ++i) {
        uint64_t selectedIdx = h_selected[i];
        // The selected index should be within [0, PopSize)
        ASSERT_LT(selectedIdx, PopSize);

        // The selected fitness should be valid
        double selectedFitness = h_population.fitnessValue[selectedIdx];
        ASSERT_GE(selectedFitness, 0.0);

        // Print the selection results
        std::cout << "Thread " << i << " selected individual " << selectedIdx << " with fitness " << selectedFitness << std::endl;
    }
}


int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    // Initialize CUDA device
    hipSetDevice(0);
    return RUN_ALL_TESTS();
}
